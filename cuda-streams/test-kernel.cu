/*

  Simple CUDA kernel test that times one kernel execution:
  * overall timing, inc. memory transfers and
  * kernel timing only.

 */

# include <stdio.h>
# include <assert.h>
# include <time.h>
# include "test-utils.h"
# include "test-utils-cuda.h"

# ifndef REAL
# define REAL float
# endif


//
// --- the MAIN
//
int main (int argc, char **argv)
{
  // total amount of memory used on GPU (1GB default)
  size_t totalBytes = NUM_GIGA;
  if (argc > 1) totalBytes = parseArgv (argc, argv);

  // sizes
  const size_t totalSize = totalBytes / sizeof(REAL);
  
  // fixed parameters
  const size_t blockSize = 256;

  // report size
  printSize (totalBytes);

  // device ID
  int devId = 0;

  // hipDeviceProp_t struct reference
  hipDeviceProp_t devProp;
  CUDA_SAFE_CALL( hipGetDeviceProperties(&devProp, devId) );
  assert (totalBytes < devProp.totalGlobalMem);
  printf(" found device %s with %fGB\n", devProp.name, ((REAL) devProp.totalGlobalMem)/((REAL) (NUM_GIGA)));
  CUDA_SAFE_CALL( hipSetDevice(devId) );

  // allocate pinned host memory and device memory
  REAL *xCPU, *yCPU, *xGPU, *yGPU;
  CUDA_SAFE_CALL( hipHostMalloc((void**) &xCPU, totalBytes) );   // host pinned
  CUDA_SAFE_CALL( hipHostMalloc((void**) &yCPU, totalBytes) );   // host pinned
  CUDA_SAFE_CALL( hipMalloc    ((void**) &xGPU, totalBytes) );   // device
  CUDA_SAFE_CALL( hipMalloc    ((void**) &yGPU, totalBytes) );   // device

  // elapsed time in milliseconds (has to be float)
  float elapsedTime, elapsedTimeAllOps;


  //
  // ..... create events and streams
  //
  hipEvent_t startEvent,       stopEvent,
              startEventAllOps, stopEventAllOps;
  CUDA_SAFE_CALL( hipEventCreate(&startEvent) );
  CUDA_SAFE_CALL( hipEventCreate(&startEventAllOps) );
  CUDA_SAFE_CALL( hipEventCreate(&stopEvent) );
  CUDA_SAFE_CALL( hipEventCreate(&stopEventAllOps) );


  //
  // ===== baseline case - sequential transfer and execute
  //
  srand(time(NULL));
  randVec <REAL> (xCPU, totalSize);
  CUDA_SAFE_CALL( hipEventRecord(startEventAllOps, 0) );
  CUDA_SAFE_CALL( hipMemcpy(xGPU, xCPU, totalBytes, hipMemcpyHostToDevice) );

  CUDA_SAFE_CALL( hipEventRecord(startEvent, 0) );
  cudaKernel <REAL> <<<totalSize/blockSize, blockSize>>> (xGPU, yGPU, totalSize, 0);
  CUDA_SAFE_CALL( hipEventRecord(stopEvent, 0) );
  CUDA_SAFE_CALL( hipEventSynchronize(stopEvent) );

  CUDA_SAFE_CALL( hipMemcpy(yCPU, yGPU, totalBytes, hipMemcpyDeviceToHost) );
  CUDA_SAFE_CALL( hipEventRecord(stopEventAllOps, 0) );
  CUDA_SAFE_CALL( hipEventSynchronize(stopEventAllOps) );

  CUDA_SAFE_CALL( hipEventElapsedTime(&elapsedTime,       startEvent,       stopEvent) );
  CUDA_SAFE_CALL( hipEventElapsedTime(&elapsedTimeAllOps, startEventAllOps, stopEventAllOps) );
  printf(" \n timing transfer and execute\n");
  printf("    ... time (overall) = %f ms\n", elapsedTimeAllOps);
  printf("    ... time (kernel)  = %f ms\n", elapsedTime);
  printf("    ... error          = %g\n", maxError <REAL> (yCPU, totalSize));


  //
  // ===== cleanup
  //
  CUDA_SAFE_CALL( hipEventDestroy(startEvent) );
  CUDA_SAFE_CALL( hipEventDestroy(startEventAllOps) );
  CUDA_SAFE_CALL( hipEventDestroy(stopEvent) );
  CUDA_SAFE_CALL( hipEventDestroy(stopEventAllOps) );

  hipFree(xGPU);
  hipFree(yGPU);
  hipHostFree(xCPU);
  hipHostFree(yCPU);

  return EXIT_SUCCESS;
}
